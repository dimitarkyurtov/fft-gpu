#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_complex.h>

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

__device__
hipFloatComplex exp(hipFloatComplex z) {
    float expReal = exp(hipCrealf(z));
    float cosImag = cos(hipCimagf(z));
    float sinImag = sin(hipCimagf(z));

    return make_hipFloatComplex(expReal * cosImag, expReal * sinImag);
}

__device__
unsigned int reverseBits(unsigned int n, unsigned int numBits)
{
    unsigned int reversed{ 0 };
    for (unsigned int i = 0; i < numBits; ++i)
    {
        unsigned int lsb { n & 1 };
        reversed = (reversed << 1) | lsb;
        n >>= 1;
    }
    return reversed;
}

__global__
void reverse_bits(const hipFloatComplex *fftSequence, hipFloatComplex *fftSequenceBitReversed, const int log2N)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  fftSequenceBitReversed[reverseBits(tid, log2N)] = make_hipFloatComplex(hipCrealf(fftSequence[tid]), hipCimagf(fftSequence[tid]));
}

__global__
void fft(hipFloatComplex *fftSequenceBitReversed, const int N, const int s)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  
  uint k { tid/(s/2) * s };
  uint j { tid%(s/2) };

  auto W{ exp(make_hipFloatComplex(0, -2 * M_PI_F * j / s)) };

  hipFloatComplex c1 { fftSequenceBitReversed[k+j] };
  hipFloatComplex c2 { fftSequenceBitReversed[k+j+s/2] };
  hipFloatComplex c3 { hipCmulf(W, c2) };

  fftSequenceBitReversed[k+j] = hipCaddf(c1, c3);
  fftSequenceBitReversed[k+j+ss/2] = hipCsubf(c1, c3);
}

void printCuComplex(hipFloatComplex c)
{
  std::cout << "(" << hipCrealf(c) << "," << hipCimagf(c) << ") ";
}

int main(void)
{
  int N = 8;
  hipFloatComplex *fftSequence, *fftSequenceBitReversed;

  checkCuda( hipMallocManaged(&fftSequence, N*sizeof(hipFloatComplex)) );
  checkCuda( hipMallocManaged(&fftSequenceBitReversed, N*sizeof(hipFloatComplex)) );

  for (size_t i = 1; i <= N; i++)
  {
    fftSequence[i-1] = make_hipFloatComplex(i, 0);
  }

  fftSequenceBitReversed[2] = make_hipFloatComplex(1,1);
  reverse_bits<<<1, 8>>>(fftSequence, fftSequenceBitReversed, log2(N));

  for (int s = 2; s <= N; s *= 2)
  {
    fft<<<1, 8>>>(fftSequenceBitReversed, N, s);
  }

  hipDeviceSynchronize();

  for (int i = 0; i < N; i++)
  {
    printCuComplex(fftSequenceBitReversed[i]);
  }
  std::cout << std::endl;

  hipFree(fftSequence);
  hipFree(fftSequenceBitReversed);
  
  return 0;
}