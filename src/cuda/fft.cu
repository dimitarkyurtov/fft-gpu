#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <hip/hip_complex.h>
#include <chrono>

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

__device__
hipFloatComplex exp(hipFloatComplex z) {
    float expReal = exp(hipCrealf(z));
    float cosImag = cos(hipCimagf(z));
    float sinImag = sin(hipCimagf(z));

    return make_hipFloatComplex(expReal * cosImag, expReal * sinImag);
}

__device__
unsigned int reverseBits(unsigned int n, unsigned int numBits)
{
    unsigned int reversed{ 0 };
    for (unsigned int i = 0; i < numBits; ++i)
    {
        unsigned int lsb { n & 1 };
        reversed = (reversed << 1) | lsb;
        n >>= 1;
    }
    return reversed;
}

__global__
void reverse_bits(const hipFloatComplex *fftSequence, hipFloatComplex *fftSequenceBitReversed, const int log2N)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  fftSequenceBitReversed[reverseBits(tid, log2N)] = make_hipFloatComplex(hipCrealf(fftSequence[tid]), hipCimagf(fftSequence[tid]));
}

__global__
void fft(hipFloatComplex *fftSequenceBitReversed, const int s)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  
  int k { tid/(s/2) * s };
  int j { tid%(s/2) };

  auto W{ exp(make_hipFloatComplex(0, -2 * M_PI * j / s)) };

  hipFloatComplex c1 { fftSequenceBitReversed[k+j] };
  hipFloatComplex c2 { fftSequenceBitReversed[k+j+s/2] };
  hipFloatComplex c3 { hipCmulf(W, c2) };

  fftSequenceBitReversed[k+j] = hipCaddf(c1, c3);
  fftSequenceBitReversed[k+j+s/2] = hipCsubf(c1, c3);
}

void printCuComplex(hipFloatComplex c)
{
  std::cout << "(" << hipCrealf(c) << "," << hipCimagf(c) << ") ";
}

int main(int argc, const char * argv[])
{
  int N = 8;

  if (argc == 2)
  {
    N = std::stoi(argv[1]);
    if (N > 100'000)
      {
          std::cerr << "Safety limit 100 000 threads.\n";
          return 1;
      }
  }
  else
  {
      std::cerr << "Usage: " << argv[0] << " <natural_number>\n";
      return 1;
  }

  auto start{ std::chrono::high_resolution_clock::now() };

  for (size_t i = 0; i < 100'00; i++)
  {
    hipFloatComplex *fftSequence, *fftSequenceBitReversed;

    checkCuda( hipMallocManaged(&fftSequence, N*sizeof(hipFloatComplex)) );
    checkCuda( hipMallocManaged(&fftSequenceBitReversed, N*sizeof(hipFloatComplex)) );

    for (size_t i = 1; i <= N; i++)
    {
      fftSequence[i-1] = make_hipFloatComplex(i, 0);
    }

    hipDeviceProp_t prop;
    int device;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    std::cout << "Maximum work group size (threads per block): " << prop.maxThreadsPerBlock << std::endl;

    /// Important to set the work group to maximum size for maximum compute unit unitilization
    int blockSize = prop.maxThreadsPerBlock > N ? N : prop.maxThreadsPerBlock;
    int numBlocks = (N + blockSize - 1) / blockSize;

    reverse_bits<<<numBlocks, blockSize>>>(fftSequence, fftSequenceBitReversed, log2(N));

    blockSize = 256 > N ? N : 256;
    numBlocks = (N/2 + blockSize - 1) / blockSize;

    for (int s = 2; s <= N; s *= 2)
    {
      fft<<<numBlocks, blockSize>>>(fftSequenceBitReversed, s);
    }

    hipDeviceSynchronize();
  }

  auto end{ std::chrono::high_resolution_clock::now() };

  std::chrono::duration<double> elapsed{ end - start };
  std::cout << "Elapsed time: " << elapsed.count() << " seconds\n";

  hipFree(fftSequence);
  hipFree(fftSequenceBitReversed);
  
  return 0;
}